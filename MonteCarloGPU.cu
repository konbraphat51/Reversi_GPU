#include <hip/hip_runtime.h>

#include "board.h"
#include "util.h"

int mcGPU_move(BoardState *state)
{
    // convert BoardState to a format that can be used by the GPU
    int board[BOARD_H * BOARD_W];
    for (int x = 0; x < BOARD_W; x++)
    {
        for (int y = 0; y < BOARD_H; y++)
        {
            board[BOARD_W * y + x] = state->board[x][y];
        }
    }
    int activePlayer = state->active_player;
    bool passed = state->passed;
}

__device__ int *get_valid_moves(int *board, int activePlayer, bool passed)
{
    int movesBuffer[BOARD_W * BOARD_H];
    int bufferIndex = 0;

    for (int x = 0; x < BOARD_W; x++)
    {
        for (int y = 0; y < BOARD_H; y++)
        {
            if (board[BOARD_W * y + x] == activePlayer)
            {
                // map_adjacent
            }
        }
    }
}

template <typename T>
__device__ void map_adjacent(const int y, const int x, const T f)
{

    if (y > 0)
    {
        f(y - 1, x);
        if (x > 0)
            f(y - 1, x - 1);
        if (x < (BOARD_W - 1))
            f(y - 1, x + 1);
    }

    if (y < (BOARD_H - 1))
    {
        f(y + 1, x);
        if (x > 0)
            f(y + 1, x - 1);
        if (x < (BOARD_W - 1))
            f(y + 1, x + 1);
    }

    if (x > 0)
        f(y, x - 1);

    if (x < (BOARD_W - 1))
        f(y, x + 1);
}